#include "hip/hip_runtime.h"
#include <iostream>

using namespace std;

#define n 100


__global__ void mul(float *d_a,float *d_b,float *d_c,int n){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;

    for(int p=0;p<n;p++){
        d_c[i][j]=d_c[i][j]+d_a[i][p]*d_b[p][j];
    }
}

int main() {
    const int s= sizeof(float)*n*n;

    int h_a[n][n]={0};
    int h_b[n][n]={0};
    int h_c[n][n]={0};

    float (*d_a)[n];
    float (*d_b)[n];
    float (*d_c)[n];

    hipMalloc((void**)&d_a,s);
    hipMalloc((void**)&d_b,s);
    hipMalloc((void**)&d_c,s);

    hipMemcpy(d_a,h_a,s,cudaMemcopyHostToDevice);
    hipMemcpy(d_b,h_b,s,cudaMemcopyHostToDevice);

    mul<<<dim3(1,1,1),dim3(n,n,1)>>>(d_a,d_b,d_c,n);

    hipMemcpy(h_c,d_c,s,cudaMemcopyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);



    return 0;
}