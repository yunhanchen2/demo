
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
using namespace std;

#define MVnum 1100

int arcs[MVnum][MVnum];

//kernal
__global__ void find_the_same(int *c,int *arr,int *n){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    if(i<j&&j<(*n)){
        if(arr[i*(*n)+j]==1){
            for(int k=j+1;k<(*n);k++){
                if(arr[k*(*n)+j]==1&&arr[i*(*n)+k]==1){
                    (*c)++;
                }
            }
        }
    }
}

struct AMGraph{
    int vexnum;
    int edgnum;
};

void init(AMGraph &G,int n1,int n2){
    G.vexnum=n1;
    G.edgnum=n2;
    //init matrix
    for(int i=0;i<G.vexnum;i++){
        for(int j=0;j<G.vexnum;j++){
            arcs[i][j]=0;
        }
    }
}

void Create(AMGraph &G,int v1,int v2){
    int i=v1-1;
    int j=v2-1;
    arcs[i][j]=1;
    arcs[j][i]=1;
}

int main() {
    //set the size to allocate
    const int arr_size=MVnum*MVnum*sizeof(int);
    const int c_size= sizeof(int);

    //init the graph and values
    AMGraph G;
    int v1,v2,n1,n2;
    ifstream fin;
    fin.open("karate_edges.txt");
    fin>>n1>>n2;
    init(G,n1,n2);
    while(fin>>v1>>v2){
        Create(G,v1,v2);
    }

    int co=0;
    int *count=&co;
    
    int n00=G.vexnum;
    int *n0=&n00;

    //declear GPU memory pointers
    int *c;
    int *arr;
    int *n;

    //allocate GPU memory
    hipMalloc((void**)&arr,arr_size);
    hipMalloc((void**)&c,c_size);
    hipMalloc((void**)&n,c_size);

    //transfer the data from CPU to GPU
    hipMemcpy(c,count,c_size,hipMemcpyHostToDevice);
    hipMemcpy(arr,arcs,arr_size,hipMemcpyHostToDevice);
    hipMemcpy(n,n0,c_size,hipMemcpyHostToDevice);

    //launch the kernal
    int m=G.vexnum/32+1;
    find_the_same<<<dim3(m,m,1),dim3(32,32,1)>>>(c,arr,n);

    //copy back the result from GPU to CPU
    hipMemcpy(count,c,c_size,hipMemcpyDeviceToHost);

    //print out the result
    cout<<"the number of triangle is "<<*count<<endl;

    //free the memory
    hipFree(c);
    hipFree(arr);
    hipFree(n);

    return 0;
}
