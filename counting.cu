
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;

struct AMGraph{
    int vexnum;
    int edgnum;
};


//initialize array
void init(AMGraph &G,int n1,int n2){
    G.vexnum=n1;
    G.edgnum=n2;
}

void get_matrix(AMGraph &G,int v1,int v2,int * matx){
    int i=v1-1;
    int j=v2-1;
    matx[i*G.vexnum+j]=1;
    matx[j*G.vexnum+i]=1;
}

__global__ void count(int *d_mark,int *matx,int *vexnum){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int ind=0;
    int *arr_tmp=new int[vexnum[0]];
    //get neighbor
    for(int j=0;j<vexnum[0];j++){
        if(matx[i*vexnum[0]+j]==1){
            arr_tmp[ind]=j;
            ind++;
        }
    }
    //count other
    d_mark[i]=0;
    for(int k=i;k<vexnum[0];k++){
        for(int p=k;p<vexnum[0];p++){
            for(int q=p;q<vexnum[0];q++){
                if(matx[k*vexnum[0]+p]+matx[k*vexnum[0]+q]+matx[p*vexnum[0]+q]==3){
                    d_mark[i]+=1;
                }
            }
        }
    }
}

int main() {
    AMGraph G;
    ifstream fin;
    fin.open("karate_edges.txt");
    int m1,m2,n1,n2;
    fin>>n1>>n2;
    init(G,n1,n2);

    //get the matrix first
    int *matx=new int[G.vexnum*G.vexnum];
    for(int i=0;i<G.vexnum*G.vexnum;i++){
        matx[i]=0;
    }
    while(fin>>m1>>m2){
        get_matrix(G,m1,m2,matx);
    }

    //counting
    int *mark=new int[G.vexnum];
    int v=G.vexnum;

    int *mark_d;
    int *vexnum;
    int *d_matx;

    hipMalloc((void**)&mark,4*G.vexnum);
    hipMalloc((void**)&vexnum,4);
    hipMalloc((void**)&d_matx,4*G.vexnum*G.vexnum);

    hipMemcpy(mark_d,mark,4*G.vexnum,hipMemcpyHostToDevice);
    hipMemcpy(vexnum,&v,4,hipMemcpyHostToDevice);
    hipMemcpy(d_matx,matx,4*G.vexnum*G.vexnum,hipMemcpyHostToDevice);

    count<<<3,1024>>>(mark_d,matx,vexnum);

    hipMemcpy(mark,mark_d,4*G.vexnum,hipMemcpyDeviceToHost);

    int sum=0;
    for(int i=0;i<G.vexnum;i++){
        sum+=mark[i];
    }

    cout<<sum<<endl;

    return 0;
}
