
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cstdlib>
using namespace std;
using namespace chrono;

__global__ void mul(float *d_a, float *d_b, float *d_c, int N){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    for(int p=0;p<N;p++){
        d_c[i*N + j] = d_c[i*N + j] + d_a[i*N + p]*d_b[p*N + j];
    }
}

int main(int argc, char * argv[]) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int n=atoi(argv[1]);

    const float s= sizeof(float)*n*n;

    float *h_a=new float[n*n];
    float *h_b=new float[n*n];
    float *h_c=new float[n*n];

    srand((unsigned)time(NULL));

    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            h_a[i*n+j]=rand();
        }
    }
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            h_b[i*n+j]=rand();
        }
    }


    float *d_a;
    float *d_b;
    float *d_c;

    hipMalloc((void**)&d_a,s);
    hipMalloc((void**)&d_b,s);
    hipMalloc((void**)&d_c,s);

    hipMemcpy(d_a,h_a,s,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,s,hipMemcpyHostToDevice);

    int m=n/1024;
    int r=n%1024;
    for(int i=0;i<m;i++){
        for(int j=0;j<1024;j++)
            mul<<<dim3(1,1,1),dim3(32,32,1)>>>(d_a,d_b,d_c,n);
    }
    for(int i=0;i<r;i++){
        mul<<<dim3(1,1,1),dim3(32,32,1)>>>(d_a,d_b,d_c,n);
    }

    hipMemcpy(h_c,d_c,s,hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << milliseconds;

    return 0;
}