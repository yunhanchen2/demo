
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

using namespace std;
using namespace chrono;

#define n 100

float h_a[n][n]={0};
float h_b[n][n]={0};
float h_c[n][n]={0};

__global__ void mul(float *d_a, float *d_b, float *d_c, int N){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    for(int p=0;p<N;p++){
        d_c[i*N + j] = d_c[i*N + j] + d_a[i*N + p]*d_b[p*N + j];
    }
}

int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    const float s= sizeof(float)*n*n;

    float *d_a;
    float *d_b;
    float *d_c;

    hipMalloc((void**)&d_a,s);
    hipMalloc((void**)&d_b,s);
    hipMalloc((void**)&d_c,s);

    hipMemcpy(d_a,h_a,s,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,s,hipMemcpyHostToDevice);

    mul<<<dim3(1,1,1),dim3(n,n,1)>>>(d_a,d_b,d_c,n);

    hipMemcpy(h_c,d_c,s,hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << milliseconds;

    return 0;
}